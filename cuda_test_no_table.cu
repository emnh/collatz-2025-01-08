#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdint>
#include <iomanip>

#define CACHE_SIZE (1ULL << 30)  // 1 GiB Cache Size
#define BASE_BITS 34             // Set to 37 for larger ranges
#define CHUNK_SIZE (1ULL << 30)  // 1 GiB of numbers per chunk
#define MAX_ITERATIONS 1024

__device__ bool is_mandatory(uint64_t nL, int base_bits) {
    __uint128_t b = static_cast<__uint128_t>(1) << base_bits; // Start with b = 2^BASE_BITS
    __uint128_t c = nL;

    while (b % 2 == 0) {
        if (b % 2 == 0 && c % 2 == 0) {
            b /= 2;
            c /= 2;
        } else if (c % 2 == 1) {
            b *= 3;
            c = 3 * c + 1;
        }

        if (b <= ((static_cast<__uint128_t>(1) << base_bits) - 1)) {
            return false;
        }
    }
    return true;
}

__device__ int count_trailing_zeros_64(uint64_t n) {
    return (n == 0) ? 64 : __ffsll(n) - 1;
}

__global__ void direct_convergence_test(uint64_t *results, uint64_t *powers_of_3, int *cache, uint64_t chunk_start, uint64_t chunk_end, int base_bits, unsigned long long *total_processed) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + chunk_start;
    if (idx >= chunk_end) return;

    // Test if the number is mandatory
    if (!is_mandatory(idx, base_bits)) return;

    // Perform convergence test
    __uint128_t n = idx;
    uint64_t n0 = static_cast<uint64_t>(n);
    int delay = 0;
    unsigned int iteration_count = 0;

    while (n > 1) {
        if (n < CACHE_SIZE && cache[static_cast<uint64_t>(n)] != -1) {
            delay += cache[static_cast<uint64_t>(n)];
            break;
        }

        if (iteration_count >= MAX_ITERATIONS) {
            printf("Exceeded maximum iterations\n");
            break;
        }

        n = n + 1;
        int a = count_trailing_zeros_64(static_cast<uint64_t>(n));
        n >>= a;
        n *= powers_of_3[a];
        n = n - 1;
        int b = count_trailing_zeros_64(static_cast<uint64_t>(n));
        n >>= b;

        delay += a + b;
        iteration_count++;
    }

    if (n0 < CACHE_SIZE) {
        cache[n0] = delay;
    }

    // Increment the total processed count
    atomicAdd(total_processed, 1);
}

void initialize_powers_of_3(uint64_t *powers_of_3_host, int max_power) {
    powers_of_3_host[0] = 1;
    for (int i = 1; i <= max_power; ++i) {
        powers_of_3_host[i] = powers_of_3_host[i - 1] * 3;
    }
}

int main() {
    const int base_bits = BASE_BITS;
    const uint64_t max_nL = static_cast<uint64_t>(1) << base_bits;
    const uint64_t chunk_size = CHUNK_SIZE;

    // Host allocations
    uint64_t *powers_of_3_host = new uint64_t[65];
    int *cache_host = new int[CACHE_SIZE];
    unsigned long long total_processed_host = 0;

    std::fill(cache_host, cache_host + CACHE_SIZE, -1);
    initialize_powers_of_3(powers_of_3_host, 64);

    // Device allocations
    uint64_t *powers_of_3_device;
    int *cache_device;
    unsigned long long *total_processed_device;

    hipMalloc(&powers_of_3_device, 65 * sizeof(uint64_t));
    hipMalloc(&cache_device, CACHE_SIZE * sizeof(int));
    hipMalloc(&total_processed_device, sizeof(unsigned long long));
    hipMemset(total_processed_device, 0, sizeof(unsigned long long));

    hipMemcpy(powers_of_3_device, powers_of_3_host, 65 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(cache_device, cache_host, CACHE_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Timer for the entire process
    auto start_total = std::chrono::high_resolution_clock::now();

    // Process in chunks
    for (uint64_t chunk_start = 0; chunk_start < max_nL; chunk_start += chunk_size) {
        uint64_t chunk_end = std::min(chunk_start + chunk_size, max_nL);

        const int threads_per_block = 256;
        const int num_blocks = ((chunk_end - chunk_start) + threads_per_block - 1) / threads_per_block;

        // Timer for chunk processing
        auto start_chunk = std::chrono::high_resolution_clock::now();

        // Launch the kernel
        direct_convergence_test<<<num_blocks, threads_per_block>>>(nullptr, powers_of_3_device, cache_device, chunk_start, chunk_end, base_bits, total_processed_device);
        hipDeviceSynchronize();

        auto end_chunk = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_chunk = end_chunk - start_chunk;

        std::cout << "Processed chunk [" << chunk_start << ", " << chunk_end << ") in "
                  << elapsed_chunk.count() << " seconds." << std::endl;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_total = end_total - start_total;

    // Copy total processed count back to host
    hipMemcpy(&total_processed_host, total_processed_device, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Output performance report
    double numbers_per_second = total_processed_host / elapsed_total.count();
    std::cout << "\nPerformance Report:\n";
    std::cout << "-------------------\n";
    std::cout << "Total processed: " << total_processed_host << " numbers\n";
    std::cout << "Total time: " << elapsed_total.count() << " seconds\n";
    std::cout << "Processing rate: " << numbers_per_second << " numbers/second\n";

    // Clean up
    hipFree(powers_of_3_device);
    hipFree(cache_device);
    hipFree(total_processed_device);

    delete[] powers_of_3_host;
    delete[] cache_host;

    return 0;
}
